
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>
#include <omp.h>

using namespace std;

const int VECTOR_SIZE = 100;

int main()
{
    // set up a random number generator
    unsigned seed = chrono::system_clock::now().time_since_epoch().count();
    mt19937 generator(seed);
    uniform_int_distribution<int> distribution(0, 9999);

    // allocate memory for the vectors
    int* vector1 = new int[VECTOR_SIZE];
    int* vector2 = new int[VECTOR_SIZE];
    int* result = new int[VECTOR_SIZE];

    // fill the vectors with random numbers
    #pragma omp parallel for
    for (int i = 0; i < VECTOR_SIZE; i++)
    {
        vector1[i] = distribution(generator);
        vector2[i] = distribution(generator);
    }

    // add the vectors in parallel using OpenMP
    #pragma omp parallel for
    for (int i = 0; i < VECTOR_SIZE; i++)
    {
        result[i] = vector1[i] + vector2[i];
    }

    // print the first and second vectors and their sum
    cout << "Vector 1: \n[";
    for (int i = 0; i < VECTOR_SIZE; i++)
    {
        cout << vector1[i];
        if (i != VECTOR_SIZE - 1)
        {
            cout << ", ";
        }
    }
    cout << "]" << endl;

    cout << "Vector 2:\n [";
    for (int i = 0; i < VECTOR_SIZE; i++)
    {
        cout << vector2[i];
        if (i != VECTOR_SIZE - 1)
        {
            cout << ", ";
        }
    }
    cout << "]" << endl;
    cout << "Result: \n[";
    for (int i = 0; i < VECTOR_SIZE; i++)
    {
        cout << result[i];
        if (i != VECTOR_SIZE - 1)
        {
            cout << ", ";
        }
    }
    cout << "]" << endl;

    // free the allocated memory
    delete[] vector1;
    delete[] vector2;
    delete[] result;

    return 0;
}



